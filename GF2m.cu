#include "hip/hip_runtime.h"
#include "GF2m.cuh"
#include <cmath>
#include <iostream>
#include <list>
#include <string>
#include <unordered_map>

uint32_t GF2m::get_high_bit(uint32_t value) {
	uint32_t result = 2147483648;

	while (value < result) {
		result >>= 1;
	}

	return result;
}

uint32_t GF2m::elements_mult(uint32_t op1, uint32_t op2) {
	/*uint32_t result = 0,
		hiBitSet;

	for (uint32_t i = 0; i < GF2m::bitsPerElement_; i++) {
		if ((op2 & 1) != 0) {
			result ^= op1;
		}
		hiBitSet = op1 & (GF2m::highBitPoly_ >> 1);
		op1 <<= 1;
		if (hiBitSet != 0) {
			op1 ^= GF2m::fieldMod_;
		}
		op2 >>= 1;
	}

	return result & (GF2m::highBitPoly_ - 1);*/
	thrust::device_vector<uint32_t> dv(bitsPerElement_);
	for (uint32_t i = 0; i < bitsPerElement_; i++) {
		dv[i] = i;
	}
	shift<uint32_t> f(op1, op2, highBitPoly_, fieldMod_);
	thrust::transform(dv.begin(), dv.end(), dv.begin(), f);
	uint32_t result = thrust::reduce(dv.begin(), dv.end(), (uint32_t)0, thrust::bit_xor<uint32_t>());

	return result;
}

std::string GF2m::element_to_bit_str(uint32_t value) {
	std::string result = "";
	uint32_t mask = highBitPoly_ >> 1;

	for (uint32_t i = 0; i < bitsPerElement_; i++) {
		result += ((mask & value) != 0) ? "1" : "0";
		mask >>= 1;
	}

	return result;
}

bool GF2m::is_element_by_value(uint32_t value) {
	for (int i = 0; i < array_size_; i++) {
		if (elements_array_[i] == value) {
			return true;
		}
	}

	return false;
}

GF2m::GF2m(uint32_t irreduciblePolynomial) {
	uint32_t primitiveElement = 2,
		fieldElement;

	highBitPoly_ = get_high_bit(irreduciblePolynomial);
	bitsPerElement_ = (uint32_t)log2(highBitPoly_);
	fieldMod_ = irreduciblePolynomial ^ highBitPoly_;
	array_size_ = highBitPoly_ - 1;

	int j = 1;
	while (j != array_size_) {
		elements_array_ = new uint32_t[array_size_];
		for (int i = 0; i < array_size_; i++) {
			elements_array_[i] = 0;
		}
		elements_array_[0] = 1;
		j = 1;

		elementsToPrint_.clear();
		elementsToPrint_.push_back(std::pair<int, uint32_t>(-1, 0));
		elementsToPrint_.push_back(std::pair<int, uint32_t>(0, 1));

		fieldElement = primitiveElement;
		for (int i = 0; i < array_size_ - 1; i++) {
			if (is_element_by_value(fieldElement)) {
				delete[] elements_array_;
				break;
			}
			elements_array_[i + 1] = fieldElement;
			elementsToPrint_.push_back(std::pair<int, uint32_t>(i + 1, fieldElement));
			fieldElement = elements_mult(fieldElement, primitiveElement);
			j++;
		}

		primitiveElement++;
	}
}

std::ostream& operator<<(std::ostream &os, GF2m& field) {
	if (field.elementsToPrint_.size() == 0) {
		os << "empty field" << std::endl;
		return os;
	}

	std::string result = "";
	for (auto element : field.elementsToPrint_) {
		if (element.first == -1) {
			result += "0 " + field.element_to_bit_str(element.second) + "\n";
		}
		else if (element.first == -1) {
			result += "1 " + field.element_to_bit_str(element.second) + "\n";
		}
		else {
			result += "a" + std::to_string(element.first) + " " + field.element_to_bit_str(element.second) + "\n";
		}
	}
	result.erase(result.length() - 1);

	os << "GF(2^" << field.bitsPerElement_ << ") is:" << std::endl << result << std::endl;

	return os;
}